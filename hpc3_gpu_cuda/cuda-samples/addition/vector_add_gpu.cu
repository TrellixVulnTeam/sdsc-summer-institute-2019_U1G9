// SDSC Summer Institute 2019
// Andreas Goetz (agoetz@sdsc.edu)

// CUDA program to add two vectors in parallel on the GPU
// launch all kernels at once
//


#include <hip/hip_runtime.h>
#include<stdio.h>

// define vector length and threads per block
#define N (255*4096)
#define TPB 512

//
// CUDA device function that adds two integer vectors
//
__global__ void add(int *a, int *b, int *c, int n){

  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  if (tid < n)
    c[tid] = a[tid] + b[tid];

}

//
// main program
//
int main(void){

  int *h_a, *h_b, *h_c;
  int *d_a, *d_b, *d_c;
  int size = N * sizeof(int);
  int i, nblock, err;

  // allocate host memory
  h_a = (int *) malloc(size);
  h_b = (int *) malloc(size);
  h_c = (int *) malloc(size);

// allocate device memory
  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);

  // initialize vectors
  for (i=0; i<N; i++){
    h_a[i] = i+1;
    h_b[i] = i+1;
  }

  // copy input data to device
  hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

  // add vectors by launching a sufficient number of blocks of the add() kernel
  nblock = (N+TPB-1)/TPB;
  printf("\nLaunching vector addition kernel...\n");
  printf("Vector length     = %d\n",N);
  printf("Blocks            = %d\n",nblock);
  printf("Threads per block = %d\n",TPB);
  printf("Kernel copies     = %d\n",nblock*TPB);
  add<<<nblock,TPB>>>(d_a, d_b, d_c, N);

  // copy results back to host
  hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

  // deallocate memory
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  // check results
  err = 0;
  for (i=0; i<N; i++){
    if (h_c[i] != 2*(i+1)) err += 1;
  }
  if (err != 0){
    printf("\n Error, %d elements do not match!\n\n", err);
  } else {
    printf("\n Success! All elements match.\n\n");
  }

  // deallocate host memory
  free(h_a);
  free(h_b);
  free(h_c);

  return err;

}
